#include <stdio.h>
#include <assert.h>

#include <hipsparse.h>

#define blockMatrixSize			3			// --- Each block of the sparse block matrix is blockMatrixSize x blockMatrixSize

/*******************/
/* iDivUp FUNCTION */
/*******************/
int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/********************/
/* CUDA ERROR CHECK */
/********************/
// --- Credit to http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) { exit(code); }
	}
}

void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }

/***************************/
/* CUSPARSE ERROR CHECKING */
/***************************/
static const char *_cusparseGetErrorEnum(hipsparseStatus_t error)
{
	switch (error)
	{

	case HIPSPARSE_STATUS_SUCCESS:
		return "HIPSPARSE_STATUS_SUCCESS";

	case HIPSPARSE_STATUS_NOT_INITIALIZED:
		return "HIPSPARSE_STATUS_NOT_INITIALIZED";

	case HIPSPARSE_STATUS_ALLOC_FAILED:
		return "HIPSPARSE_STATUS_ALLOC_FAILED";

	case HIPSPARSE_STATUS_INVALID_VALUE:
		return "HIPSPARSE_STATUS_INVALID_VALUE";

	case HIPSPARSE_STATUS_ARCH_MISMATCH:
		return "HIPSPARSE_STATUS_ARCH_MISMATCH";

	case HIPSPARSE_STATUS_MAPPING_ERROR:
		return "HIPSPARSE_STATUS_MAPPING_ERROR";

	case HIPSPARSE_STATUS_EXECUTION_FAILED:
		return "HIPSPARSE_STATUS_EXECUTION_FAILED";

	case HIPSPARSE_STATUS_INTERNAL_ERROR:
		return "HIPSPARSE_STATUS_INTERNAL_ERROR";

	case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
		return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

	case HIPSPARSE_STATUS_ZERO_PIVOT:
		return "HIPSPARSE_STATUS_ZERO_PIVOT";
	}

	return "<unknown>";
}

inline void __cusparseSafeCall(hipsparseStatus_t err, const char *file, const int line)
{
	if (HIPSPARSE_STATUS_SUCCESS != err) {
		fprintf(stderr, "CUSPARSE error in file '%s', line %d, error %s\nterminating!\n", __FILE__, __LINE__, \
			_cusparseGetErrorEnum(err)); \
			assert(0); \
	}
}

extern "C" void cusparseSafeCall(hipsparseStatus_t err) { __cusparseSafeCall(err, __FILE__, __LINE__); }

/********/
/* MAIN */
/********/
int main() {

	// --- Initialize cuSPARSE
	hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	// --- Initialize matrix descriptors
	hipsparseMatDescr_t descrA, descrC;
	cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	cusparseSafeCall(hipsparseCreateMatDescr(&descrC));

	const int M = 9;						// --- Number of rows
	const int N = 9;						// --- Number of columns

	const int nnzb = 7;						// --- Number of non-zero blocks

	const int Mb = iDivUp(M, blockMatrixSize);
	const int Nb = iDivUp(N, blockMatrixSize);

	float h_Ab0[blockMatrixSize * blockMatrixSize] = { 4.f, -1.f, 0.f, -1.f, 4.f, -1.f, 0.f, -1.f, 4.f };
	float h_Ab1[blockMatrixSize * blockMatrixSize] = { -1.f, 0.f, 0.f, 0.f, -1.f, 0.f, 0.f, 0.f, -1.f };

	// --- Host vectors defining the block-sparse matrix
	float *h_bsrValA = (float *)malloc(blockMatrixSize * blockMatrixSize * nnzb * sizeof(float));
	int *h_bsrRowPtrA = (int *)malloc((Mb + 1) * sizeof(int));
	int *h_bsrColIndA = (int *)malloc(nnzb * sizeof(int));

	memcpy(h_bsrValA, h_Ab0, blockMatrixSize * blockMatrixSize * sizeof(float));
	memcpy(h_bsrValA + 1 * blockMatrixSize * blockMatrixSize, h_Ab1, blockMatrixSize * blockMatrixSize 
		* sizeof(float));
	memcpy(h_bsrValA + 2 * blockMatrixSize * blockMatrixSize, h_Ab1, blockMatrixSize * blockMatrixSize 
		* sizeof(float));
	memcpy(h_bsrValA + 3 * blockMatrixSize * blockMatrixSize, h_Ab0, blockMatrixSize * blockMatrixSize 
		* sizeof(float));
	memcpy(h_bsrValA + 4 * blockMatrixSize * blockMatrixSize, h_Ab1, blockMatrixSize * blockMatrixSize 
		* sizeof(float));
	memcpy(h_bsrValA + 5 * blockMatrixSize * blockMatrixSize, h_Ab1, blockMatrixSize * blockMatrixSize 
		* sizeof(float));
	memcpy(h_bsrValA + 6 * blockMatrixSize * blockMatrixSize, h_Ab0, blockMatrixSize * blockMatrixSize 
		* sizeof(float));

	h_bsrRowPtrA[0] = 0;	h_bsrRowPtrA[1] = 2;	h_bsrRowPtrA[2] = 5;	h_bsrRowPtrA[3] = 7;

	h_bsrColIndA[0] = 0;	h_bsrColIndA[1] = 1;	h_bsrColIndA[2] = 0;	h_bsrColIndA[3] = 1;
	h_bsrColIndA[4] = 2;	h_bsrColIndA[5] = 1;	h_bsrColIndA[6] = 2;

	// --- Device vectors defining the block-sparse matrix
	float *d_bsrValA;		gpuErrchk(hipMalloc(&d_bsrValA, blockMatrixSize * blockMatrixSize 
		* nnzb * sizeof(float)));
	int *d_bsrRowPtrA;		gpuErrchk(hipMalloc(&d_bsrRowPtrA, (Mb + 1) * sizeof(int)));
	int *d_bsrColIndA;		gpuErrchk(hipMalloc(&d_bsrColIndA, nnzb * sizeof(int)));

	gpuErrchk(hipMemcpy(d_bsrValA, h_bsrValA, blockMatrixSize * blockMatrixSize * nnzb * sizeof(float), 
		hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_bsrRowPtrA, h_bsrRowPtrA, (Mb + 1) * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_bsrColIndA, h_bsrColIndA, nnzb * sizeof(int), hipMemcpyHostToDevice));

	// --- Transforming bsr to csr format
	hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;
	const int nnz = nnzb * blockMatrixSize * blockMatrixSize; // --- Number of non-zero elements
	int *d_csrRowPtrC;		gpuErrchk(hipMalloc(&d_csrRowPtrC, (M + 1) * sizeof(int)));
	int *d_csrColIndC;		gpuErrchk(hipMalloc(&d_csrColIndC, nnz		* sizeof(int)));
	float *d_csrValC;		gpuErrchk(hipMalloc(&d_csrValC, nnz		* sizeof(float)));
	cusparseSafeCall(hipsparseSbsr2csr(handle, dir, Mb, Nb, descrA, d_bsrValA, d_bsrRowPtrA, d_bsrColIndA, 
		blockMatrixSize, descrC, d_csrValC, d_csrRowPtrC, d_csrColIndC));

	// --- Transforming csr to dense format
	float *d_A;				gpuErrchk(hipMalloc(&d_A, M * N * sizeof(float)));
	cusparseSafeCall(hipsparseScsr2dense(handle, M, N, descrC, d_csrValC, d_csrRowPtrC, d_csrColIndC, d_A, M));

	float *h_A = (float *)malloc(M * N * sizeof(float));
	gpuErrchk(hipMemcpy(h_A, d_A, M * N * sizeof(float), hipMemcpyDeviceToHost));
	
	// --- m is row index, n column index
	for (int m = 0; m < M; m++) {
		for (int n = 0; n < N; n++) {
			printf("%f ", h_A[m + n * M]);
		}
		printf("\n");
	}

	return 0;
}
