#include <stdio.h>
#include <assert.h>

#include <hipsparse.h>

/*******************/
/* iDivUp FUNCTION */
/*******************/
int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/********************/
/* CUDA ERROR CHECK */
/********************/
// --- Credit to http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) { exit(code); }
	}
}

void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }

/***************************/
/* CUSPARSE ERROR CHECKING */
/***************************/
static const char *_cusparseGetErrorEnum(hipsparseStatus_t error)
{
	switch (error)
	{

	case HIPSPARSE_STATUS_SUCCESS:
		return "HIPSPARSE_STATUS_SUCCESS";

	case HIPSPARSE_STATUS_NOT_INITIALIZED:
		return "HIPSPARSE_STATUS_NOT_INITIALIZED";

	case HIPSPARSE_STATUS_ALLOC_FAILED:
		return "HIPSPARSE_STATUS_ALLOC_FAILED";

	case HIPSPARSE_STATUS_INVALID_VALUE:
		return "HIPSPARSE_STATUS_INVALID_VALUE";

	case HIPSPARSE_STATUS_ARCH_MISMATCH:
		return "HIPSPARSE_STATUS_ARCH_MISMATCH";

	case HIPSPARSE_STATUS_MAPPING_ERROR:
		return "HIPSPARSE_STATUS_MAPPING_ERROR";

	case HIPSPARSE_STATUS_EXECUTION_FAILED:
		return "HIPSPARSE_STATUS_EXECUTION_FAILED";

	case HIPSPARSE_STATUS_INTERNAL_ERROR:
		return "HIPSPARSE_STATUS_INTERNAL_ERROR";

	case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
		return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

	case HIPSPARSE_STATUS_ZERO_PIVOT:
		return "HIPSPARSE_STATUS_ZERO_PIVOT";
	}

	return "<unknown>";
}

inline void __cusparseSafeCall(hipsparseStatus_t err, const char *file, const int line)
{
	if (HIPSPARSE_STATUS_SUCCESS != err) {
		fprintf(stderr, "CUSPARSE error in file '%s', line %d, error %s\nterminating!\n", __FILE__, __LINE__, \
			_cusparseGetErrorEnum(err)); \
			assert(0); \
	}
}

extern "C" void cusparseSafeCall(hipsparseStatus_t err) { __cusparseSafeCall(err, __FILE__, __LINE__); }

/*****************************/
/* SETUP DESCRIPTOR FUNCTION */
/*****************************/
void setUpDescriptor(hipsparseMatDescr_t &descrA, hipsparseMatrixType_t matrixType, hipsparseIndexBase_t indexBase) {
	cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	cusparseSafeCall(hipsparseSetMatType(descrA, matrixType));
	cusparseSafeCall(hipsparseSetMatIndexBase(descrA, indexBase));
}

/********************************************************/
/* DENSE TO SPARSE CONVERSION FOR REAL DOUBLE PRECISION */
/********************************************************/
void dense2SparseD(const double * __restrict__ d_A_dense, int **d_nnzPerVector, double **d_A,
	int **d_A_RowIndices, int **d_A_ColIndices, int &nnz, hipsparseMatDescr_t descrA,
	const hipsparseHandle_t handle, const int Nrows, const int Ncols) {

	const int lda = Nrows;                      // --- Leading dimension of dense matrix

	gpuErrchk(hipMalloc(&d_nnzPerVector[0], Nrows * sizeof(int)));

	// --- Compute the number of nonzero elements per row and the total number of nonzero elements 
	//     the dense d_A_dense
	cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, Nrows, Ncols, descrA, d_A_dense,
		lda, d_nnzPerVector[0], &nnz));

	// --- Device side sparse matrix
	gpuErrchk(hipMalloc(&d_A[0], nnz * sizeof(double)));
	gpuErrchk(hipMalloc(&d_A_RowIndices[0], (Nrows + 1) * sizeof(int)));
	gpuErrchk(hipMalloc(&d_A_ColIndices[0], nnz * sizeof(int)));

	cusparseSafeCall(hipsparseDdense2csr(handle, Nrows, Ncols, descrA, d_A_dense, lda, d_nnzPerVector[0],
		d_A[0], d_A_RowIndices[0], d_A_ColIndices[0]));
}

/********/
/* MAIN */
/********/
int main() {

	hipsparseHandle_t	handle;

	// --- Initialize cuSPARSE
	cusparseSafeCall(hipsparseCreate(&handle));

	// --- Initialize matrix descriptors
	hipsparseMatDescr_t descrA;
	setUpDescriptor(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ONE);

	/**************************/
	/* SETTING UP THE PROBLEM */
	/**************************/
	const int Nrows = 5;                        // --- Number of rows
	const int Ncols = 4;                        // --- Number of columns
	const int N = Nrows;

	// --- Host side dense matrix
	double *h_A_dense = (double*)malloc(Nrows * Ncols * sizeof(*h_A_dense));

	// --- Column-major storage
	h_A_dense[0] = 0.4612;  h_A_dense[5] = 0.0;       h_A_dense[10] = 1.3;     h_A_dense[15] = 0.0;
	h_A_dense[1] = 0.0;     h_A_dense[6] = 1.443;     h_A_dense[11] = 0.0;     h_A_dense[16] = 0.0;
	h_A_dense[2] = -0.0006; h_A_dense[7] = 0.4640;    h_A_dense[12] = 0.0723;  h_A_dense[17] = 0.0;
	h_A_dense[3] = 0.3566;  h_A_dense[8] = 0.0;       h_A_dense[13] = 0.7543;  h_A_dense[18] = 0.0;
	h_A_dense[4] = 0.;      h_A_dense[9] = 0.0;       h_A_dense[14] = 0.0;     h_A_dense[19] = 0.1;

	// --- Create device array and copy host array to it
	double *d_A_dense;  gpuErrchk(hipMalloc(&d_A_dense, Nrows * Ncols * sizeof(double)));
	gpuErrchk(hipMemcpy(d_A_dense, h_A_dense, Nrows * Ncols * sizeof(*d_A_dense), hipMemcpyHostToDevice));

	/*******************************/
	/* FROM DENSE TO SPARSE MATRIX */
	/*******************************/
	int nnz = 0;            // --- Number of nonzero elements in dense matrix
	int *d_nnzPerVector;	// --- Device side number of nonzero elements per row

	double *d_A;		// --- Sparse matrix values - array of size nnz
	int *d_A_RowIndices;	// --- "Row indices"
	int *d_A_ColIndices;	// --- "Column indices"

	dense2SparseD(d_A_dense, &d_nnzPerVector, &d_A, &d_A_RowIndices, &d_A_ColIndices, nnz, descrA,
		handle, Nrows, Ncols);

	/*******************************************************/
	/* CHECKING THE RESULTS FOR DENSE TO SPARSE CONVERSION */
	/*******************************************************/
	// --- Host side number of nonzero elements per row
	int *h_nnzPerVector = (int *)malloc(Nrows * sizeof(int));
	gpuErrchk(hipMemcpy(h_nnzPerVector, d_nnzPerVector, Nrows * sizeof(int), hipMemcpyDeviceToHost));

	printf("Number of nonzero elements in dense matrix = %i\n\n", nnz);
	for (int i = 0; i < Nrows; ++i)
		printf("Number of nonzero elements in row %i = %i \n", i, h_nnzPerVector[i]);
	printf("\n");

	// --- Host side sparse matrix
	double *h_A = (double *)malloc(nnz * sizeof(double));
	int *h_A_RowIndices = (int *)malloc((Nrows + 1) * sizeof(int));
	int *h_A_ColIndices = (int *)malloc(nnz * sizeof(int));
	gpuErrchk(hipMemcpy(h_A, d_A, nnz * sizeof(double), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_A_RowIndices, d_A_RowIndices, (Nrows + 1) * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_A_ColIndices, d_A_ColIndices, nnz * sizeof(int), hipMemcpyDeviceToHost));

	printf("\nOriginal matrix in CSR format\n\n");
	for (int i = 0; i < nnz; ++i) printf("A[%i] = %f\n", i, h_A[i]);
	printf("\n\n");

	for (int i = 0; i < (Nrows + 1); ++i) printf("h_A_RowIndices[%i] = %i \n", i, h_A_RowIndices[i]);
	printf("\n");
	for (int i = 0; i < nnz; ++i) printf("h_A_ColIndices[%i] = %i \n", i, h_A_ColIndices[i]);

	/*******************************/
	/* FROM SPARSE TO DENSE MATRIX */
	/*******************************/
	double *d_A_denseReconstructed; gpuErrchk(hipMalloc(&d_A_denseReconstructed,
		Nrows * Ncols * sizeof(double)));
	cusparseSafeCall(hipsparseDcsr2dense(handle, Nrows, Ncols, descrA, d_A, d_A_RowIndices, d_A_ColIndices,
		d_A_denseReconstructed, Nrows));

	/*******************************************************/
	/* CHECKING THE RESULTS FOR SPARSE TO DENSE CONVERSION */
	/*******************************************************/
	double *h_A_denseReconstructed = (double *)malloc(Nrows * Ncols * sizeof(double));
	gpuErrchk(hipMemcpy(h_A_denseReconstructed, d_A_denseReconstructed, Nrows * Ncols * sizeof(double),
		hipMemcpyDeviceToHost));

	printf("\nReconstructed dense matrix \n");
	for (int m = 0; m < Nrows; m++) {
		for (int n = 0; n < Ncols; n++)
			printf("%f\t", h_A_denseReconstructed[n * Nrows + m]);
		printf("\n");
	}

	return 0;
}
