#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>

#include "Utilities.cuh"

#include <hip/hip_runtime.h>
#include <hipsparse.h>

/********/
/* MAIN */
/********/
int main()
{
	// --- Initialize cuSPARSE
	hipsparseHandle_t handle;	cusparseSafeCall(hipsparseCreate(&handle));

	/**************************/
	/* SETTING UP THE PROBLEM */
	/**************************/
	const int N     = 4;				// --- Number of rows and columns

	// --- Host side dense matrices
	double *h_A_dense = (double*)malloc(N * N * sizeof(*h_A_dense));
	double *h_B_dense = (double*)malloc(N * N * sizeof(*h_B_dense));
	double *h_C_dense = (double*)malloc(N * N * sizeof(*h_C_dense));
	
	// --- Column-major ordering
	h_A_dense[0] = 0.4612;  h_A_dense[4] = -0.0006;		h_A_dense[8]  = 0.3566;		h_A_dense[12] = 0.0; 
	h_A_dense[1] = -0.0006; h_A_dense[5] = 0.4640;		h_A_dense[9]  = 0.0723;		h_A_dense[13] = 0.0; 
	h_A_dense[2] = 0.3566;  h_A_dense[6] = 0.0723;		h_A_dense[10] = 0.7543;		h_A_dense[14] = 0.0; 
	h_A_dense[3] = 0.;		h_A_dense[7] = 0.0;			h_A_dense[11] = 0.0;		h_A_dense[15] = 0.1; 

	// --- Column-major ordering
	h_B_dense[0] = 0.;		h_B_dense[4] = 0.;			h_B_dense[8]  = 1.;			h_B_dense[12] = 0.; 
	h_B_dense[1] = 1.;		h_B_dense[5] = 0.;			h_B_dense[9]  = 0.;			h_B_dense[13] = 0.; 
	h_B_dense[2] = 0.;		h_B_dense[6] = 1.;			h_B_dense[10] = 0.;			h_B_dense[14] = 0.; 
	h_B_dense[3] = 0.;		h_B_dense[7] = 0.;			h_B_dense[11] = 0.;			h_B_dense[15] = 1.; 

	// --- Create device arrays and copy host arrays to them
	double *d_A_dense;	gpuErrchk(hipMalloc(&d_A_dense, N * N * sizeof(*d_A_dense)));
	double *d_B_dense;	gpuErrchk(hipMalloc(&d_B_dense, N * N * sizeof(*d_B_dense)));
	double *d_C_dense;	gpuErrchk(hipMalloc(&d_C_dense, N * N * sizeof(*d_C_dense)));
	gpuErrchk(hipMemcpy(d_A_dense, h_A_dense, N * N * sizeof(*d_A_dense), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_B_dense, h_B_dense, N * N * sizeof(*d_B_dense), hipMemcpyHostToDevice));
	
	// --- Descriptor for sparse matrix A
	hipsparseMatDescr_t descrA;		cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	cusparseSafeCall(hipsparseSetMatType		(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
	cusparseSafeCall(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));  
	
	// --- Descriptor for sparse matrix B
	hipsparseMatDescr_t descrB;		cusparseSafeCall(hipsparseCreateMatDescr(&descrB));
	cusparseSafeCall(hipsparseSetMatType		(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL));
	cusparseSafeCall(hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ONE));  

	// --- Descriptor for sparse matrix C
	hipsparseMatDescr_t descrC;		cusparseSafeCall(hipsparseCreateMatDescr(&descrC));
	cusparseSafeCall(hipsparseSetMatType		(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL));
	cusparseSafeCall(hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ONE));  

	int nnzA = 0;							// --- Number of nonzero elements in dense matrix A
	int nnzB = 0;							// --- Number of nonzero elements in dense matrix B

	const int lda = N;						// --- Leading dimension of dense matrix

	// --- Device side number of nonzero elements per row of matrix A
	int *d_nnzPerVectorA; 	gpuErrchk(hipMalloc(&d_nnzPerVectorA, N * sizeof(*d_nnzPerVectorA)));
	cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, N, N, descrA, d_A_dense, lda, d_nnzPerVectorA, &nnzA));

	// --- Device side number of nonzero elements per row of matrix B
	int *d_nnzPerVectorB; 	gpuErrchk(hipMalloc(&d_nnzPerVectorB, N * sizeof(*d_nnzPerVectorB)));
	cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, N, N, descrB, d_B_dense, lda, d_nnzPerVectorB, &nnzB));

	// --- Host side number of nonzero elements per row of matrix A
	int *h_nnzPerVectorA = (int *)malloc(N * sizeof(*h_nnzPerVectorA));
	gpuErrchk(hipMemcpy(h_nnzPerVectorA, d_nnzPerVectorA, N * sizeof(*h_nnzPerVectorA), hipMemcpyDeviceToHost));

	// --- Host side number of nonzero elements per row of matrix B
	int *h_nnzPerVectorB = (int *)malloc(N * sizeof(*h_nnzPerVectorB));
	gpuErrchk(hipMemcpy(h_nnzPerVectorB, d_nnzPerVectorB, N * sizeof(*h_nnzPerVectorB), hipMemcpyDeviceToHost));

	printf("Number of nonzero elements in dense matrix A = %i\n\n", nnzA);
	for (int i = 0; i < N; ++i) printf("Number of nonzero elements in row %i for matrix = %i \n", i, h_nnzPerVectorA[i]);
	printf("\n");

	printf("Number of nonzero elements in dense matrix B = %i\n\n", nnzB);
	for (int i = 0; i < N; ++i) printf("Number of nonzero elements in row %i for matrix = %i \n", i, h_nnzPerVectorB[i]);
	printf("\n");

	// --- Device side sparse matrix
	double *d_A;			gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
	double *d_B;			gpuErrchk(hipMalloc(&d_B, nnzB * sizeof(*d_B)));
	
	int *d_A_RowIndices;	gpuErrchk(hipMalloc(&d_A_RowIndices, (N + 1) * sizeof(*d_A_RowIndices)));
	int *d_B_RowIndices;	gpuErrchk(hipMalloc(&d_B_RowIndices, (N + 1) * sizeof(*d_B_RowIndices)));
	int *d_C_RowIndices;	gpuErrchk(hipMalloc(&d_C_RowIndices, (N + 1) * sizeof(*d_C_RowIndices)));
	int *d_A_ColIndices;	gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
	int *d_B_ColIndices;	gpuErrchk(hipMalloc(&d_B_ColIndices, nnzB * sizeof(*d_B_ColIndices)));
	
	cusparseSafeCall(hipsparseDdense2csr(handle, N, N, descrA, d_A_dense, lda, d_nnzPerVectorA, d_A, d_A_RowIndices, d_A_ColIndices));
	cusparseSafeCall(hipsparseDdense2csr(handle, N, N, descrB, d_B_dense, lda, d_nnzPerVectorB, d_B, d_B_RowIndices, d_B_ColIndices));

	// --- Host side sparse matrices
	double *h_A = (double *)malloc(nnzA * sizeof(*h_A));		
	double *h_B = (double *)malloc(nnzB * sizeof(*h_B));		
	int *h_A_RowIndices = (int *)malloc((N + 1) * sizeof(*h_A_RowIndices));
	int *h_A_ColIndices = (int *)malloc(nnzA * sizeof(*h_A_ColIndices));
	int *h_B_RowIndices = (int *)malloc((N + 1) * sizeof(*h_B_RowIndices));
	int *h_B_ColIndices = (int *)malloc(nnzB * sizeof(*h_B_ColIndices));
	int *h_C_RowIndices = (int *)malloc((N + 1) * sizeof(*h_C_RowIndices));
	gpuErrchk(hipMemcpy(h_A, d_A, nnzA * sizeof(*h_A), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_A_RowIndices, d_A_RowIndices, (N + 1) * sizeof(*h_A_RowIndices), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_A_ColIndices, d_A_ColIndices, nnzA * sizeof(*h_A_ColIndices), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_B, d_B, nnzB * sizeof(*h_B), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_B_RowIndices, d_B_RowIndices, (N + 1) * sizeof(*h_B_RowIndices), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_B_ColIndices, d_B_ColIndices, nnzB * sizeof(*h_B_ColIndices), hipMemcpyDeviceToHost));
	
	printf("\nOriginal matrix A in CSR format\n\n");
	for (int i = 0; i < nnzA; ++i) printf("A[%i] = %f ", i, h_A[i]); printf("\n");

	printf("\nOriginal matrix B in CSR format\n\n");
	for (int i = 0; i < nnzB; ++i) printf("B[%i] = %f ", i, h_B[i]); printf("\n");

	printf("\n");
	for (int i = 0; i < (N + 1); ++i) printf("h_A_RowIndices[%i] = %i \n", i, h_A_RowIndices[i]); printf("\n");

	printf("\n");
	for (int i = 0; i < (N + 1); ++i) printf("h_B_RowIndices[%i] = %i \n", i, h_B_RowIndices[i]); printf("\n");

	printf("\n");
	for (int i = 0; i < nnzA; ++i) printf("h_A_ColIndices[%i] = %i \n", i, h_A_ColIndices[i]);	
	
	printf("\n");
	for (int i = 0; i < nnzB; ++i) printf("h_B_ColIndices[%i] = %i \n", i, h_B_ColIndices[i]);	

	// --- Performing the matrix - matrix multiplication
	int baseC, nnzC = 0;
	// nnzTotalDevHostPtr points to host memory
	int *nnzTotalDevHostPtr = &nnzC;	
	
	cusparseSafeCall(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

	cusparseSafeCall(hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, N, descrB, nnzB, 
										 d_B_RowIndices, d_B_ColIndices, descrA, nnzA, d_A_RowIndices, d_A_ColIndices, descrC, d_C_RowIndices, 
										 nnzTotalDevHostPtr));
	if (NULL != nnzTotalDevHostPtr) nnzC = *nnzTotalDevHostPtr;
	else {
		gpuErrchk(hipMemcpy(&nnzC,  d_C_RowIndices + N, sizeof(int), hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(&baseC, d_C_RowIndices,     sizeof(int), hipMemcpyDeviceToHost));
		nnzC -= baseC;
	}
	int *d_C_ColIndices;	gpuErrchk(hipMalloc(&d_C_ColIndices, nnzC * sizeof(int)));
	double *d_C;			gpuErrchk(hipMalloc(&d_C, nnzC * sizeof(double)));
	double *h_C = (double *)malloc(nnzC * sizeof(*h_C));		
	int *h_C_ColIndices = (int *)malloc(nnzC * sizeof(*h_C_ColIndices));
	cusparseSafeCall(hipsparseDcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, N, descrB, nnzB,
									  d_B, d_B_RowIndices, d_B_ColIndices, descrA, nnzA, d_A, d_A_RowIndices, d_A_ColIndices, descrC,
									  d_C, d_C_RowIndices, d_C_ColIndices));

	cusparseSafeCall(hipsparseDcsr2dense(handle, N, N, descrC, d_C, d_C_RowIndices, d_C_ColIndices, d_C_dense, N));

	gpuErrchk(hipMemcpy(h_C ,           d_C,            nnzC * sizeof(*h_C), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_C_RowIndices, d_C_RowIndices, (N + 1) * sizeof(*h_C_RowIndices), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_C_ColIndices, d_C_ColIndices, nnzC * sizeof(*h_C_ColIndices), hipMemcpyDeviceToHost));
	
	printf("\nResult matrix C in CSR format\n\n");
	for (int i = 0; i < nnzC; ++i) printf("C[%i] = %f ", i, h_C[i]); printf("\n");

	printf("\n");
	for (int i = 0; i < (N + 1); ++i) printf("h_C_RowIndices[%i] = %i \n", i, h_C_RowIndices[i]); printf("\n");

	printf("\n");
	for (int i = 0; i < nnzC; ++i) printf("h_C_ColIndices[%i] = %i \n", i, h_C_ColIndices[i]);	
	
	gpuErrchk(hipMemcpy(h_C_dense, d_C_dense, N * N * sizeof(double), hipMemcpyDeviceToHost));

	for (int j = 0; j < N; j++) {
		for (int i = 0; i < N; i++) 
			printf("%f \t", h_C_dense[i * N + j]);
		printf("\n");
		}
}
