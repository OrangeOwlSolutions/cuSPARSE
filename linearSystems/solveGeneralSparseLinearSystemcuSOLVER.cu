#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hipsparse.h>
#include <hipsolver.h>

/*******************/
/* iDivUp FUNCTION */
/*******************/
//extern "C" int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }
__host__ __device__ int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/********************/
/* CUDA ERROR CHECK */
/********************/
// --- Credit to http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) { exit(code); }
	}
}

extern "C" void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }

/**************************/
/* CUSOLVE ERROR CHECKING */
/**************************/
static const char *_cusolverGetErrorEnum(hipsolverStatus_t error)
{
	switch (error)
	{
	case HIPSOLVER_STATUS_SUCCESS:
		return "CUSOLVER_SUCCESS";

	case HIPSOLVER_STATUS_NOT_INITIALIZED:
		return "HIPSOLVER_STATUS_NOT_INITIALIZED";

	case HIPSOLVER_STATUS_ALLOC_FAILED:
		return "HIPSOLVER_STATUS_ALLOC_FAILED";

	case HIPSOLVER_STATUS_INVALID_VALUE:
		return "HIPSOLVER_STATUS_INVALID_VALUE";

	case HIPSOLVER_STATUS_ARCH_MISMATCH:
		return "HIPSOLVER_STATUS_ARCH_MISMATCH";

	case HIPSOLVER_STATUS_EXECUTION_FAILED:
		return "HIPSOLVER_STATUS_EXECUTION_FAILED";

	case HIPSOLVER_STATUS_INTERNAL_ERROR:
		return "HIPSOLVER_STATUS_INTERNAL_ERROR";

	case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
		return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

	}

	return "<unknown>";
}

inline void __cusolveSafeCall(hipsolverStatus_t err, const char *file, const int line)
{
	if (HIPSOLVER_STATUS_SUCCESS != err) {
		fprintf(stderr, "CUSOLVE error in file '%s', line %d, error: %s \nterminating!\n", __FILE__, __LINE__, \
			_cusolverGetErrorEnum(err)); \
			assert(0); \
	}
}

extern "C" void cusolveSafeCall(hipsolverStatus_t err) { __cusolveSafeCall(err, __FILE__, __LINE__); }

/***************************/
/* CUSPARSE ERROR CHECKING */
/***************************/
static const char *_cusparseGetErrorEnum(hipsparseStatus_t error)
{
	switch (error)
	{

	case HIPSPARSE_STATUS_SUCCESS:
		return "HIPSPARSE_STATUS_SUCCESS";

	case HIPSPARSE_STATUS_NOT_INITIALIZED:
		return "HIPSPARSE_STATUS_NOT_INITIALIZED";

	case HIPSPARSE_STATUS_ALLOC_FAILED:
		return "HIPSPARSE_STATUS_ALLOC_FAILED";

	case HIPSPARSE_STATUS_INVALID_VALUE:
		return "HIPSPARSE_STATUS_INVALID_VALUE";

	case HIPSPARSE_STATUS_ARCH_MISMATCH:
		return "HIPSPARSE_STATUS_ARCH_MISMATCH";

	case HIPSPARSE_STATUS_MAPPING_ERROR:
		return "HIPSPARSE_STATUS_MAPPING_ERROR";

	case HIPSPARSE_STATUS_EXECUTION_FAILED:
		return "HIPSPARSE_STATUS_EXECUTION_FAILED";

	case HIPSPARSE_STATUS_INTERNAL_ERROR:
		return "HIPSPARSE_STATUS_INTERNAL_ERROR";

	case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
		return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

	case HIPSPARSE_STATUS_ZERO_PIVOT:
		return "HIPSPARSE_STATUS_ZERO_PIVOT";
	}

	return "<unknown>";
}

inline void __cusparseSafeCall(hipsparseStatus_t err, const char *file, const int line)
{
	if (HIPSPARSE_STATUS_SUCCESS != err) {
		fprintf(stderr, "CUSPARSE error in file '%s', line %Ndims\Nobjs %s\nerror %Ndims: %s\nterminating!\Nobjs", __FILE__, __LINE__, err, \
			_cusparseGetErrorEnum(err)); \
			hipDeviceReset(); assert(0); \
	}
}

extern "C" void cusparseSafeCall(hipsparseStatus_t err) { __cusparseSafeCall(err, __FILE__, __LINE__); }

/********/
/* MAIN */
/********/
int main()
{
	// --- Initialize cuSPARSE
	hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

	const int Nrows = 4;                        // --- Number of rows
	const int Ncols = 4;                        // --- Number of columns
	const int N = Nrows;

	// --- Host side dense matrix
	double *h_A_dense = (double*)malloc(Nrows*Ncols*sizeof(*h_A_dense));

	// --- Column-major ordering
	h_A_dense[0] = 1.0f; h_A_dense[4] = 4.0f; h_A_dense[8] = 0.0f; h_A_dense[12] = 0.0f;
	h_A_dense[1] = 0.0f; h_A_dense[5] = 2.0f; h_A_dense[9] = 3.0f; h_A_dense[13] = 0.0f;
	h_A_dense[2] = 5.0f; h_A_dense[6] = 0.0f; h_A_dense[10] = 0.0f; h_A_dense[14] = 7.0f;
	h_A_dense[3] = 0.0f; h_A_dense[7] = 0.0f; h_A_dense[11] = 9.0f; h_A_dense[15] = 0.0f;

	//create device array and copy host to it
	double *d_A_dense;  gpuErrchk(hipMalloc(&d_A_dense, Nrows * Ncols * sizeof(*d_A_dense)));
	gpuErrchk(hipMemcpy(d_A_dense, h_A_dense, Nrows * Ncols * sizeof(*d_A_dense), hipMemcpyHostToDevice));

	// --- Descriptor for sparse matrix A
	hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

	int nnz = 0;                                // --- Number of nonzero elements in dense matrix
	const int lda = Nrows;                      // --- Leading dimension of dense matrix
	// --- Device side number of nonzero elements per row
	int *d_nnzPerVector;    gpuErrchk(hipMalloc(&d_nnzPerVector, Nrows * sizeof(*d_nnzPerVector)));
	cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, Nrows, Ncols, descrA, d_A_dense, lda, d_nnzPerVector, &nnz));
	// --- Host side number of nonzero elements per row
	int *h_nnzPerVector = (int *)malloc(Nrows * sizeof(*h_nnzPerVector));
	gpuErrchk(hipMemcpy(h_nnzPerVector, d_nnzPerVector, Nrows * sizeof(*h_nnzPerVector), hipMemcpyDeviceToHost));

	printf("Number of nonzero elements in dense matrix = %i\n\n", nnz);
	for (int i = 0; i < Nrows; ++i) printf("Number of nonzero elements in row %i = %i \n", i, h_nnzPerVector[i]);
	printf("\n");

	// --- Device side dense matrix
	double *d_A;            gpuErrchk(hipMalloc(&d_A, nnz * sizeof(*d_A)));
	int *d_A_RowIndices;    gpuErrchk(hipMalloc(&d_A_RowIndices, (Nrows + 1) * sizeof(*d_A_RowIndices)));
	int *d_A_ColIndices;    gpuErrchk(hipMalloc(&d_A_ColIndices, nnz * sizeof(*d_A_ColIndices)));

	cusparseSafeCall(hipsparseDdense2csr(handle, Nrows, Ncols, descrA, d_A_dense, lda, d_nnzPerVector, d_A, d_A_RowIndices, d_A_ColIndices));

	// --- Host side dense matrix
	double *h_A = (double *)malloc(nnz * sizeof(*h_A));
	int *h_A_RowIndices = (int *)malloc((Nrows + 1) * sizeof(*h_A_RowIndices));
	int *h_A_ColIndices = (int *)malloc(nnz * sizeof(*h_A_ColIndices));
	gpuErrchk(hipMemcpy(h_A, d_A, nnz*sizeof(*h_A), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_A_RowIndices, d_A_RowIndices, (Nrows + 1) * sizeof(*h_A_RowIndices), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_A_ColIndices, d_A_ColIndices, nnz * sizeof(*h_A_ColIndices), hipMemcpyDeviceToHost));

	for (int i = 0; i < nnz; ++i) printf("A[%i] = %.0f ", i, h_A[i]); printf("\n");

	for (int i = 0; i < (Nrows + 1); ++i) printf("h_A_RowIndices[%i] = %i \n", i, h_A_RowIndices[i]); printf("\n");

	for (int i = 0; i < nnz; ++i) printf("h_A_ColIndices[%i] = %i \n", i, h_A_ColIndices[i]);

	// --- Allocating and defining dense host and device data vectors
	double *h_y = (double *)malloc(Nrows * sizeof(double));
	h_y[0] = 100.0;  h_y[1] = 200.0; h_y[2] = 400.0; h_y[3] = 500.0;

	double *d_y;        gpuErrchk(hipMalloc(&d_y, Nrows * sizeof(double)));
	gpuErrchk(hipMemcpy(d_y, h_y, Nrows * sizeof(double), hipMemcpyHostToDevice));

	// --- Allocating the host and device side result vector
	double *h_x = (double *)malloc(Ncols * sizeof(double));
	double *d_x;        gpuErrchk(hipMalloc(&d_x, Ncols * sizeof(double)));

	// --- CUDA solver initialization
	hipsolverSpHandle_t solver_handle;
	hipsolverSpCreate(&solver_handle);

	// --- Using LU factorization
	//int singularity;
	//cusolveSafeCall(cusolverSpDcsrlsvluHost(solver_handle, N, nnz, descrA, h_A, h_A_RowIndices, h_A_ColIndices, h_y, 0.000001, 0, h_x, &singularity));
	// --- Using QR factorization
	//int singularity;
	//cusolveSafeCall(cusolverSpDcsrlsvqrHost(solver_handle, N, nnz, descrA, h_A, h_A_RowIndices, h_A_ColIndices, h_y, 0.000001, 0, h_x, &singularity));

	int rankA;
	int *p = (int *)malloc(N * sizeof(int));
	double min_norm;
	cusolveSafeCall(cusolverSpDcsrlsqvqrHost(solver_handle, N, N, nnz, descrA, h_A, h_A_RowIndices, h_A_ColIndices, h_y, 0.000001, &rankA, h_x, p, &min_norm));

	printf("Showing the results...\n");
	for (int i = 0; i < N; i++) printf("%f\n", h_x[i]);

}
